
#include <hip/hip_runtime.h>
__global__ void SineWave_kernel(float4 *pos, unsigned int meshWidth, unsigned int meshHeight, float animationTime){
	
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x/(float)meshWidth;
	float v = y/(float)meshHeight;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;
	float frequency = 4.0f;

	float w = sinf(u*frequency + animationTime)*cosf(v*frequency + animationTime)*0.5f;

	pos[y * meshWidth + x] = make_float4(u, w ,v, 1.0f);

}

void launchCUDAkernel(float4 *pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(meshWidth/block.x, meshHeight/block.y, 1);

	SineWave_kernel<<<grid,block>>>(pos, meshWidth, meshHeight, time);


}